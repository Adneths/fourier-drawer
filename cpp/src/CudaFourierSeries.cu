#include "hip/hip_runtime.h"
#include "CudaFourierSeries.cuh"

#include "core.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <cuda_gl_interop.h>

#include <iostream>

__inline__ __device__ float2 warpAllReduceSum(float2 val) {
#pragma unroll
	for (int mask = warpSize / 2; mask > 0; mask /= 2)
	{
		val.x += __shfl_xor(val.x, mask);
		val.y += __shfl_xor(val.y, mask);
	}
	return val;
}
__inline__ __device__ float2 blockReduceSum(float2 val, int resultantWarp) {

	static __shared__ float2 shared[32];
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	val = warpAllReduceSum(val);
	if (lane == 0) shared[wid] = val;

	__syncthreads();

	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : make_float2(0,0);
	if (wid == resultantWarp) val = warpAllReduceSum(val);

	return val;
}

__global__ void sumVector(float* mags, float* pathPtr, size_t size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size)
		return;

	float2 v = { mags[id * 2], mags[id * 2 + 1] };
	float2 val = blockReduceSum(v, 0);
	if (id == 0)
	{
		atomicAdd(&pathPtr[id * 2], val.x);
		atomicAdd(&pathPtr[id * 2 + 1], val.y);
	}
}

__global__ void cudaIncrement(float* mags, int* freqs, float* pathCache, size_t size, float dt, size_t count)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	//int lane = threadIdx.x % warpSize;
	if (id >= size)
		return;

	float2 v = { mags[id * 2], mags[id * 2 + 1] };
	float2 s = { cos(dt * freqs[id]), sin(dt * freqs[id]) };

	float2 psum = make_float2(0,0);
	for (int i = 0; i < count; i++)
	{
		v = { v.x * s.x - v.y * s.y, v.x * s.y + v.y * s.x };
		float2 val = blockReduceSum(v, i >> 5);
		if (id == i)
			psum = val;
	}
	mags[id * 2] = v.x;
	mags[id * 2 + 1] = v.y;

	if (id < count)
	{
		atomicAdd(&pathCache[id * 2], psum.x);
		atomicAdd(&pathCache[id * 2 + 1], psum.y);
	}
}

#define INCREMENT_BLOCK_SIZE 1024
CudaFourierSeries::CudaFourierSeries(LineStrip* vectorLine, Lines* pathLine, std::complex<float>* mags, int* freqs, size_t size, float dt, size_t cacheSize)
	: vectorLine(vectorLine), pathLine(pathLine), dt(dt), cacheSize(cacheSize), size(size), time(0), head(0)
{
	hipMalloc(&deviceMags, sizeof(float) * size * 2ull);
	hipMemcpy(deviceMags, (float*)mags, sizeof(float) * size * 2ull, hipMemcpyHostToDevice);
	hipMalloc(&deviceFreqs, sizeof(int) * size);
	hipMemcpy(deviceFreqs, freqs, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMalloc(&devicePathCache, sizeof(float) * cacheSize * 2ull);
	hipGraphicsGLRegisterBuffer(&vectorPtr, vectorLine->getBuffer(), hipGraphicsRegisterFlagsWriteDiscard);
	hipGraphicsGLRegisterBuffer(&pathPtr, pathLine->getBuffer(), hipGraphicsRegisterFlagsNone);

	float* deviceStart;
	hipMalloc(&deviceStart, sizeof(float) * 2ull);
	sumVector<<<(size + INCREMENT_BLOCK_SIZE - 1) / INCREMENT_BLOCK_SIZE, INCREMENT_BLOCK_SIZE>>>
			(deviceMags, deviceStart, size);
	float hostStart[3] = { 0 };
	hipMemcpy(hostStart, deviceStart, sizeof(float) * 2, hipMemcpyDeviceToHost);
	if (pathLine->isTimestamped())
		glClearBufferData(GL_ARRAY_BUFFER, GL_RGB32F, GL_RGBA, GL_FLOAT, &hostStart);
	else
		glClearBufferData(GL_ARRAY_BUFFER, GL_RG32F, GL_RGBA, GL_FLOAT, &hostStart);

	lineWidth = (pathLine->isTimestamped() ? 6ull : 4ull);
	pathBufferSize = pathLine->getCount() * lineWidth;
}
CudaFourierSeries::~CudaFourierSeries()
{
	hipFree(deviceMags);
	hipFree(deviceFreqs);
	hipFree(devicePathCache);
	hipGraphicsUnregisterResource(vectorPtr);
	hipGraphicsUnregisterResource(pathPtr);
}

float CudaFourierSeries::increment(size_t count, float time)
{
	hipMemset(devicePathCache, 0, sizeof(float) * cacheSize * 2ull);
	cudaIncrement<<<(size + INCREMENT_BLOCK_SIZE - 1) / INCREMENT_BLOCK_SIZE, INCREMENT_BLOCK_SIZE>>>
			(deviceMags, deviceFreqs, devicePathCache, size, dt, count);
	this->time += count * dt;
	return count * dt;
}

__global__ void fillVector(float* mags, size_t len, float* vector)
{

}
__global__ void fillPath(float* pathCache, size_t cacheLen, float* path, size_t pathLen, size_t head)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id * 2 >= cacheLen)
		return;
	
	int index = (head + id * 4 + 2) % pathLen;
	path[index] = pathCache[id * 2];
	path[index + 1] = pathCache[id * 2 + 1];

	index = (index + 2) % pathLen;
	path[index] = pathCache[id * 2];
	path[index + 1] = pathCache[id * 2 + 1];
	
	if (id * 2 == cacheLen - 2)
	{
		index = (index + 2) % pathLen;
		path[index] = pathCache[id * 2];
		path[index + 1] = pathCache[id * 2 + 1];
	}
}
__global__ void fillPathTimestamped(float* pathCache, size_t cacheLen, float* path, size_t pathLen, float time, float dt, size_t head)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id * 2 >= cacheLen)
		return;

	int index = (head + id * 6 + 3) % pathLen;
	path[index] = pathCache[id * 2];
	path[index + 1] = pathCache[id * 2 + 1];
	path[index + 2] = time + id * dt;

	index = (index + 3) % pathLen;
	path[index] = pathCache[id * 2];
	path[index + 1] = pathCache[id * 2 + 1];
	path[index + 2] = time + id * dt;
	
	if (id * 2 == cacheLen - 2)
	{
		index = (index + 3) % pathLen;
		path[index] = pathCache[id * 2];
		path[index + 1] = pathCache[id * 2 + 1];
		path[index + 2] = time + id * dt;
	}
}
#define CACHE_BLOCK_SIZE 64
void CudaFourierSeries::updateBuffers()
{
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	float* ptr;
	size_t mappedSize = (size + 1) * 2 * sizeof(float);
	/*hipGraphicsMapResources(1, &vectorPtr);
	hipGraphicsResourceGetMappedPointer((void**)&ptr, &mappedSize, vectorPtr);
	hipGraphicsUnmapResources(1, &vectorPtr);
	*/

	mappedSize = sizeof(float) * lineWidth * pathLine->getCount();
	hipGraphicsMapResources(1, &pathPtr);
	hipGraphicsResourceGetMappedPointer((void**)&ptr, &mappedSize, pathPtr);
	if (pathLine->isTimestamped())
		fillPathTimestamped<<<(cacheSize + CACHE_BLOCK_SIZE - 1) / CACHE_BLOCK_SIZE, CACHE_BLOCK_SIZE>>>
		(devicePathCache, cacheSize * 2, ptr, mappedSize, time - dt * cacheSize, dt, head);
	else
		fillPath<<<(cacheSize + CACHE_BLOCK_SIZE - 1) / CACHE_BLOCK_SIZE, CACHE_BLOCK_SIZE >>>
		(devicePathCache, cacheSize * 2, ptr, mappedSize, head);
	hipGraphicsUnmapResources(1, &pathPtr);

	head = (head + lineWidth * cacheSize) % (pathBufferSize);
}

void CudaFourierSeries::readyBuffers()
{
	hipDeviceSynchronize();
}

DLL_API FourierSeries* __cdecl instantiate(LineStrip* vectorLine, Lines* pathLine, std::complex<float>* mags, int* freqs, size_t size, float dt, size_t cacheSize)
{
	return new CudaFourierSeries(vectorLine, pathLine, mags, freqs, size, dt, cacheSize);
}
