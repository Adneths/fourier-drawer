#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "constant.h"
#include <iostream>

extern "C" {
	DLL_API int __cdecl printGPUInfo()
	{
		hipError_t err;
		int devicesCount;
		if (err = hipGetDeviceCount(&devicesCount))
		{
			printf("%s\n", hipGetErrorString(err));
			return -1;
		}
		int n = 0;
		printf("Avaliable GPUs\n");
		for (int i = 0; i < devicesCount; ++i)
		{
			hipDeviceProp_t deviceProperties;
			if (err = hipGetDeviceProperties(&deviceProperties, i))
			{
				printf("%s\n", hipGetErrorString(err));
				return -1;
			}
			if (deviceProperties.major >= CUDA_MINIMUM_MAJOR_VERSION
				&& deviceProperties.minor >= CUDA_MINIMUM_MINOR_VERSION)
				printf("[%d] %s - Compute Capability %d.%d\n", n++, deviceProperties.name, deviceProperties.major, deviceProperties.minor);
		}

		return 0;
	}
}